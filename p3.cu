
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 32
#define BLOCK_SIZE 32

__global__ void add_kernel(int *X, int *Y, int *Z){
    int i = threadIdx.x;
    int j = threadIdx.y;

    int index = i*N+j;

    Z[index] = X[index] + Y[index];
}

int main()
{
    int n;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    printf("input positive integer n: ");
    scanf("%d", &n);
    

    int X[N*N];
    int Y[N*N];

    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            X[i*N+j] = 0;
            Y[i*N+j] = 1;
        }
    }

    int Z[N*N];

    int *d_X, *d_Y, *d_Z;
    hipMalloc((void**) &d_X, (N*N)*sizeof(int));
    hipMalloc((void**) &d_Y, (N*N)*sizeof(int));
    hipMalloc((void**) &d_Z, (N*N)*sizeof(int));

    hipMemcpy(d_X, &X, (N*N)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, &Y, (N*N)*sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(2, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    hipEventRecord(start);
    add_kernel<<<dimGrid, dimBlock>>>(d_X, d_Y, d_Z);
    hipEventRecord(stop);

    hipMemcpy(&Z, d_Z, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_Z);

    printf("%f ms\n", milliseconds);
    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){
            printf("%d ", Z[i*N+j]);
        }
        printf("\n");
    }
}